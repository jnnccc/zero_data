#include "hip/hip_runtime.h"
#include <limits>
#include <stdio.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/transform.h>
#include <thrust/sequence.h>
#include <thrust/transform_reduce.h>
#include <algorithm>
#include <iostream>
#include <cmath>

template <typename T>
struct square
{
	__attribute__((always_inline))
	__host__ __device__
	T operator()(const T& x) const
	{ 
		return x * x;
	}
};

struct saxpy
{
	const double a;

	saxpy(double a_) : a(a_) { }

	__attribute__((always_inline))
	__host__ __device__
	double operator()(const double& x, const double& y) const
	{
		return a * x + y;
	}
};

struct saxpb
{
	const double a, b;
	
	saxpb(double a_, double b_) : a(a_), b(b_) { }

	__attribute__((always_inline))
	__host__ __device__
	double operator()(const double& x) const
	{
		return a * x + b;
	}	
};

void saxpy_fast(double A, thrust::device_vector<double>& X, thrust::device_vector<double>& Y)
{
	// Y <- A * X + Y
	thrust::transform(X.begin(), X.end(), Y.begin(), Y.begin(), saxpy(A));
}
//序列相乘
struct xy_functor : public thrust::binary_function<double,double,double>
{
	__host__ __device__
		double operator()(const double& x, const double& y) const { 
			return x * y;
		}
};
void xy_fast(thrust::device_vector<double>& X, thrust::device_vector<double>& Y)
{
	// Y <- A * X + Y
	thrust::transform(X.begin(), X.end(), Y.begin(), Y.begin(), xy_functor());
}

//cos函数
struct cos_func {

__host__ __device__
  double operator()(double x){
	return cos(x);
  }
};
//序列平方
struct t2_func {
__host__ __device__
  double operator()(double x){
	return x*x;
  }
};
//序列立方
struct t3_func {

__host__ __device__
  double operator()(double x){
	return x*x*x;
  }
};

using namespace std;

int main(void)
{
	int n = 1000000;
	double t_factor = 1.0 / n;
	double c0 = 1.0, c1 = 2.0, c2 = 3.0, c3 = 4.0, c4 = 1.0, c5 = 6.0;

	thrust::host_vector<double> h_t(n);
	thrust::device_vector<double> d_c0(n);
	thrust::device_vector<double> d_amp(n);
	thrust::device_vector<double> d_s0(n);
	thrust::device_vector<double> d_t(n);
	thrust::device_vector<double> d_t2(n);
	thrust::device_vector<double> d_t3(n);

	// 序列生成 (time series)
	thrust::sequence(d_t.begin(), d_t.end());
	thrust::transform(d_t.begin(), d_t.end(), d_t2.begin(), d_t2.begin(), saxpy(t_factor));
	d_t = d_t2;

	// 仿真序列(simulation signal)
	for (int i = 0; i < n; ++i)
		h_t[i] = rand() / (double)RAND_MAX;

	thrust::device_vector<double> d_s = h_t;
	
	clock_t t1 = clock();
	{
		// t^2序列
		thrust::transform(d_t.begin(), d_t.end(), d_t2.begin(), t2_func());

		// t^3序列
		thrust::transform(d_t.begin(), d_t.end(), d_t3.begin(), t3_func());

		// 线性操作1
		thrust::transform(d_t.begin(), d_t.end(), d_c0.begin(), saxpb(c1, c0));

		// 线性操作2
		saxpy_fast(c2, d_t2, d_c0);

		// 线性操作3
		saxpy_fast(c3, d_t3, d_c0);

		// 三角函数	
		thrust::transform(d_c0.begin(), d_c0.end(), d_s0.begin(), cos_func());

		// 幅度函数
		thrust::transform(d_t.begin(), d_t.end(), d_amp.begin(), saxpb(c5, c4));

		// 形成函数
		xy_fast(d_amp, d_s0);

		// O-C
		saxpy_fast(-1.0, d_s0, d_s);
	}

	double norm;
	clock_t t2 = clock();
	{
		// 平方 (这里把平方和求和分开了)
		norm = sqrt(thrust::transform_reduce(
			d_s.begin(), d_s.end(), square<double>(), 0.0, thrust::plus<double>()));
	}
	clock_t t3 = clock();

	cout << "平方 (linear opration) : " << (double)(t2 - t1) * 1000.0 / CLOCKS_PER_SEC << " ms" << endl;
	cout << "求和 (reduction) : " << (double)(t3 - t2) * 1000.0 / CLOCKS_PER_SEC << " ms" << endl;
	cout.precision(std::numeric_limits<double>::max_digits10 + 1);
	cout << "范数 (norm) : " << norm << endl;

	return 0;
}

